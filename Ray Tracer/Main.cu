#include "hip/hip_runtime.h"
#include <SFML/Graphics.hpp>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "Vec.cuh"
#include "VBO.h"
#include <cmath>


typedef Vec3<float> Vec3f;
typedef Vec4<float> Vec4f;
typedef VBO<float> VBOf;
typedef d_VBO<float> d_VBOf;

const unsigned int INITIALWIDTH = 600;
const unsigned int INITIALHEIGHT = 600;
unsigned int WIDTH = INITIALWIDTH;
unsigned int HEIGHT = INITIALHEIGHT;

sf::Texture screen;

__device__ bool Clear(float r, float g, float b, float a, sf::Uint8* ColorBuffer, int N) {
	for (int i = 0; i < N * 4; i += 4) {
		ColorBuffer[i + 0] = r;
		ColorBuffer[i + 1] = g;
		ColorBuffer[i + 2] = b;
		ColorBuffer[i + 3] = a;
	}
	return true;
}

__device__ Vec3f NormalOfTri(Vec3f a, Vec3f b, Vec3f c) {
	return (a - c).Cross(a - b);
}
template <typename T>
__device__ float4 area(const Vec3<T> &a, const Vec3<T> &b, const Vec3<T> &c) {
	return abs(a[1] * (b[2] - c[2]) + b[1] * (c[2] - a[2]) + c[1] * (a[2] - b[2])) / 2;
}

__device__ bool Intersect(Vec3f Pos, Vec3f Vec, float &t, d_VBOf* d_vbo) {
	return true;
}

__global__ void Render(sf::Uint8 *ColorBuffer, int WIDTH, int HEIGHT, d_VBOf *d_vbo) {
	float4 t = INFINITY;
	if(Intersect(Vec3f(), Vec3f(0,0,1), t, d_vbo))Clear(d_vbo->Color[0], d_vbo->Color[1], d_vbo->Color[2], d_vbo->Color[3], ColorBuffer, WIDTH * HEIGHT);
	else Clear(255,0,0,255,ColorBuffer,WIDTH * HEIGHT);

}


sf::Uint8* ColorBuffer;

int main() {
	//setup sf variables
	screen.create(WIDTH, HEIGHT);
	sf::RenderWindow window(sf::VideoMode(WIDTH, HEIGHT), "Ray Tracer", sf::Style::Close | sf::Style::Resize);
	sf::Sprite mSprite;
	mSprite.setTexture(screen);
	sf::Event evnt;

	sf::Uint8* ColorBuffer, *d_ColorBuffer;

	ColorBuffer = new sf::Uint8[WIDTH * HEIGHT * 4];

	for(int i = 0; i < WIDTH * HEIGHT * 4; i++)
	ColorBuffer[i] = 0;

	hipMalloc(&d_ColorBuffer, sizeof(sf::Uint8) * WIDTH * HEIGHT * 4);

	
	Vec3f tri[3] = { Vec3f(0,0,2),Vec3f(0,200,2),Vec3f(200,0,2) };

	VBOf vbo;
	vbo.addIncices(0);
	vbo.addIncices(1);
	vbo.addIncices(2);
	vbo.addVec(tri[0]);
	vbo.addVec(tri[1]);
	vbo.addVec(tri[2]);
	vbo.Color = Vec4f(0,255,0,255);

	d_VBOf *d_vbo;

	d_VBOf host_vbo (vbo);

	hipMalloc(&d_vbo, sizeof(d_VBOf));

	hipMemcpy(d_vbo, &host_vbo, sizeof(host_vbo), hipMemcpyHostToDevice);

	while (window.isOpen()) {

		while (window.pollEvent(evnt)) {
			switch (evnt.type) {
			case sf::Event::Closed:
				window.close();
				break;
			case sf::Event::KeyPressed:
				switch (evnt.key.code) {
				case sf::Keyboard::Escape:
					window.close();
					break;
				}
				break;
			case sf::Event::Resized:
				if (window.getSize().x < 600) {
					window.setSize(sf::Vector2u(INITIALWIDTH, HEIGHT));
					break;
				}
				if (window.getSize().y < 600) {
					window.setSize(sf::Vector2u(WIDTH, INITIALHEIGHT));
					break;
				}


				WIDTH = window.getSize().x;
				HEIGHT = window.getSize().y;
				hipFree(ColorBuffer);
				hipMalloc(&ColorBuffer, sizeof(sf::Uint8) * WIDTH * HEIGHT * 4);

					break;
			}
		}

		sf::Clock clock;
		//render

		Render <<<1, 1 >> >(d_ColorBuffer, WIDTH, HEIGHT, d_vbo);
		
		hipMemcpy(ColorBuffer, d_ColorBuffer, sizeof(sf::Uint8) * WIDTH * HEIGHT * 4, hipMemcpyDeviceToHost);
		
		//push render to screen
		screen.update(ColorBuffer);

		window.clear(sf::Color::Black);

		window.draw(mSprite);

		window.display();

		std::cout << clock.restart().asMilliseconds() << std::endl;
	}
	hipFree(d_ColorBuffer);
	return 0;
}

