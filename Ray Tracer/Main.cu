#include "hip/hip_runtime.h"
#include <SFML/Graphics.hpp>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "Vec.cuh"
#include "VBO.h"
#include <cmath>


typedef Vec3<float> Vec3f;
typedef Vec4<float> Vec4f;
typedef VBO<float> VBOf;

const unsigned int INITIALWIDTH = 600;
const unsigned int INITIALHEIGHT = 600;
unsigned int WIDTH = INITIALWIDTH;
unsigned int HEIGHT = INITIALHEIGHT;
unsigned int FOV = 90;


sf::Texture screen;

__device__ void Clear(float r, float g, float b, float a, sf::Uint8* ColorBuffer, int N) {
	for (int i = 0; i < N * 4; i += 4) {
		ColorBuffer[i + 0] = r;
		ColorBuffer[i + 1] = g;
		ColorBuffer[i + 2] = b;
		ColorBuffer[i + 3] = a;
	}
}



__global__ void Render(sf::Uint8 *ColorBuffer, int WIDTH, int HEIGHT, std::vector<VBOf> VBOs) {
	Clear(255,0,0,255, ColorBuffer, WIDTH * HEIGHT);
	//generate camera rays
	//camera pos = 0,0,0
	//camera vec = 1,0,0
	for(int y = 0; y < HEIGHT; y++){
		for (int x = 0; x < WIDTH; x++) {
			float t;
			for(VBOf vbo: VBOs){
				if (Intersect(Vec3f(), Vec3f(1, (2 * x - 1) * (WIDTH / HEIGHT) * tan(FOV / 2), (1 - 2 * y) * tan(FOV / 2)).Normalize(), t, vbo)) {
					ColorBuffer[x + (y * WIDTH)] = vbo.Color[0];
					ColorBuffer[x + (y * WIDTH) + 1] = vbo.Color[1];
					ColorBuffer[x + (y * WIDTH) + 2] = vbo.Color[2];
					ColorBuffer[x + (y * WIDTH) + 3] = vbo.Color[3];
				}
			}
		}
	}
}
	
__device__ float area(Vec3f a, Vec3f b, Vec3f c) {
	return abs(a[1] * (b[2] - c[2]) + b[1] * (c[2] - a[2]) + c[1] * (a[2] - b[2])) / 2;
}

__device__ bool Intersect(Vec3f Pos, Vec3f Vec, float t, VBOf vbo) {
	for(int i = 0; i <3; i++){
	if(area(vbo.vertices.at(vbo.indices.at(i))) == )
	}
}
int main() {
	//setup sf variables
	screen.create(WIDTH, HEIGHT);
	sf::RenderWindow window(sf::VideoMode(WIDTH, HEIGHT), "Ray Tracer", sf::Style::Close | sf::Style::Resize);
	sf::Sprite mSprite;
	mSprite.setTexture(screen);
	sf::Event evnt;



	sf::Uint8* ColorBuffer, *d_ColorBuffer;

	ColorBuffer = new sf::Uint8[WIDTH * HEIGHT * 4];

	for(int i = 0; i < WIDTH * HEIGHT * 4; i++)
	ColorBuffer[i] = 0;

	hipMalloc(&d_ColorBuffer, sizeof(sf::Uint8) * WIDTH * HEIGHT * 4);

	
	Vec3f tri[3] = { Vec3f(0,0,2),Vec3f(0,200,2),Vec3f(200,0,2) };
	
	VBOf vbo;
	vbo.addVec(tri[0]);
	vbo.addVec(tri[1]);
	vbo.addVec(tri[2]);
	vbo.addIncices(0);
	vbo.addIncices(1);
	vbo.addIncices(2);
	vbo.Color = Vec4f(0,0,255,255);
	
	std::vector<VBOf> objects;
	objects.push_back(vbo);

	while (window.isOpen()) {

		while (window.pollEvent(evnt)) {
			switch (evnt.type) {
			case sf::Event::Closed:
				window.close();
				break;
			case sf::Event::KeyPressed:
				switch (evnt.key.code) {
				case sf::Keyboard::Escape:
					window.close();
					break;
				}
				break;
			case sf::Event::Resized:
				if (window.getSize().x < 600) {
					window.setSize(sf::Vector2u(INITIALWIDTH, HEIGHT));
					break;
				}
				if (window.getSize().y < 600) {
					window.setSize(sf::Vector2u(WIDTH, INITIALHEIGHT));
					break;
				}


				WIDTH = window.getSize().x;
				HEIGHT = window.getSize().y;
				hipFree(d_ColorBuffer);
				hipMalloc(&d_ColorBuffer, sizeof(sf::Uint8) * WIDTH * HEIGHT * 4);
				delete ColorBuffer;
				ColorBuffer = new sf::Uint8[WIDTH * HEIGHT * 4];
					break;
			}
		}

		sf::Clock clock;
		//render
		
		Render <<<1, 1 >>>(d_ColorBuffer, WIDTH, HEIGHT, objects);
		
		hipMemcpy(ColorBuffer, d_ColorBuffer, sizeof(sf::Uint8) * WIDTH * HEIGHT * 4, hipMemcpyDeviceToHost);
		
		//push render to screen
		screen.update(ColorBuffer);

		window.draw(mSprite);

		window.display();

		std::cout << clock.restart().asMilliseconds() << std::endl;
	}
	hipFree(d_ColorBuffer);
	return 0;
}

