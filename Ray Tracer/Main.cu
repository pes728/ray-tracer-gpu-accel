#include "hip/hip_runtime.h"
#include <SFML/Graphics.hpp>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "Vec.cuh"
#include "VBO.h"
#include <cmath>


typedef Vec3<float> Vec3f;
typedef Vec4<float> Vec4f;
typedef VBO<float> VBOf;

const unsigned int INITIALWIDTH = 600;
const unsigned int INITIALHEIGHT = 600;
unsigned int WIDTH = INITIALWIDTH;
unsigned int HEIGHT = INITIALHEIGHT;

sf::Texture screen;

__device__ void Clear(float r, float g, float b, float a, sf::Uint8* ColorBuffer, int N) {
	for (int i = 0; i < N * 4; i += 4) {
		ColorBuffer[i + 0] = r;
		ColorBuffer[i + 1] = g;
		ColorBuffer[i + 2] = b;
		ColorBuffer[i + 3] = a;
	}
}

__global__ void Render(sf::Uint8 *ColorBuffer, int WIDTH, int HEIGHT, std::vector<VBOf> VBOs) {
	Clear(255,0,0,255, ColorBuffer, N);
	for(int y = 0; y <HEIGHT; y++){
		for(int x = 0; x < WIDTH; x++){
			float t;
			for(VBOf vbo:VBOs){
				//cameraPos = 0,0,0
				//cameraVec = 0,0,1
				if(Intersect(Vec3f(), Vec3f(0,0,1) , &t,vbo)){
					ColorBuffer[x + (y * WIDTH)] = vbo.color[0];
					ColorBuffer[x + (y * WIDTH) + 1] = vbo.color[1];
					ColorBuffer[x + (y * WIDTH) + 2] = vbo.color[2];
					ColorBuffer[x + (y * WIDTH) + 3] = vbo.color[3];
				}
			}
		}
	}
	
	__device__ float area(Vec3f a, Vec3f b, Vec3f c){
		return abs(a[1] * (b[2] - c[2]) + b[1] * (c[2] - a[2]) +  c[1] * (a[2] - b[2]))/2;
	}
	
	__device__ bool Intersect(Vec3f Pos, Vec3f Vec, float t, VBOf vbo)){
		if(area())
	}

sf::Uint8* ColorBuffer;

int main() {
	//setup sf variables
	screen.create(WIDTH, HEIGHT);
	sf::RenderWindow window(sf::VideoMode(WIDTH, HEIGHT), "Ray Tracer", sf::Style::Close | sf::Style::Resize);
	sf::Sprite mSprite;
	mSprite.setTexture(screen);
	sf::Event evnt;

	sf::Uint8* ColorBuffer, *d_ColorBuffer;

	ColorBuffer = new sf::Uint8[WIDTH * HEIGHT * 4];

	for(int i = 0; i < WIDTH * HEIGHT * 4; i++)
	ColorBuffer[i] = 0;

	hipMalloc(&d_ColorBuffer, sizeof(sf::Uint8) * WIDTH * HEIGHT * 4);

	
	Vec3f tri[3] = { Vec3f(0,0,2),Vec3f(0,200,2),Vec3f(200,0,2) };
	
	VBOf vbo;
	vbo.addVec(tri[0]);
	vbo.addVec(tri[1]);
	vbo.addVec(tri[2]);
	vbo.addIndices(0);
	vbo.addIndices(1);
	vbo.addIndices(2);
	vbo.color = Vec4f(0,0,255,255);
	
	std::vector<VBOf> objects;
	objects.push_back(vbo);

	while (window.isOpen()) {

		while (window.pollEvent(evnt)) {
			switch (evnt.type) {
			case sf::Event::Closed:
				window.close();
				break;
			case sf::Event::KeyPressed:
				switch (evnt.key.code) {
				case sf::Keyboard::Escape:
					window.close();
					break;
				}
				break;
			case sf::Event::Resized:
				if (window.getSize().x < 600) {
					window.setSize(sf::Vector2u(INITIALWIDTH, HEIGHT));
					break;
				}
				if (window.getSize().y < 600) {
					window.setSize(sf::Vector2u(WIDTH, INITIALHEIGHT));
					break;
				}


				WIDTH = window.getSize().x;
				HEIGHT = window.getSize().y;
				hipFree(ColorBuffer);
				hipMalloc(&ColorBuffer, sizeof(sf::Uint8) * WIDTH * HEIGHT * 4);

					break;
			}
		}

		sf::Clock clock;
		//render

		Render <<<1, 1 >> >(d_ColorBuffer, WIDTH, HEIGHT, objects);
		
		hipMemcpy(ColorBuffer, d_ColorBuffer, sizeof(sf::Uint8) * WIDTH * HEIGHT * 4, hipMemcpyDeviceToHost);
		
		//push render to screen
		screen.update(ColorBuffer);

		window.draw(mSprite);

		window.display();

		std::cout << clock.restart().asMilliseconds() << std::endl;
	}
	hipFree(d_ColorBuffer);
	return 0;
}

