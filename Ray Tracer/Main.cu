#include "hip/hip_runtime.h"
#include <SFML/Graphics.hpp>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "Vec.cuh"
#include "VBO.h"
#include <cmath>


typedef Vec3<float> Vec3f;
typedef Vec4<float> Vec4f;
typedef VBO<float> VBOf;

const unsigned int INITIALWIDTH = 600;
const unsigned int INITIALHEIGHT = 600;
unsigned int WIDTH = INITIALWIDTH;
unsigned int HEIGHT = INITIALHEIGHT;

sf::Texture screen;

__device__ void Clear(float r, float g, float b, float a, sf::Uint8* ColorBuffer, int N) {
	for (int i = 0; i < N * 4; i += 4) {
		ColorBuffer[i + 0] = r;
		ColorBuffer[i + 1] = g;
		ColorBuffer[i + 2] = b;
		ColorBuffer[i + 3] = a;
	}
}

__global__ void Render(sf::Uint8 *ColorBuffer, int N) {
	Clear(255,0,0,255, ColorBuffer, N);

sf::Uint8* ColorBuffer;

int main() {
	//setup sf variables
	screen.create(WIDTH, HEIGHT);
	sf::RenderWindow window(sf::VideoMode(WIDTH, HEIGHT), "Ray Tracer", sf::Style::Close | sf::Style::Resize);
	sf::Sprite mSprite;
	mSprite.setTexture(screen);
	sf::Event evnt;

	sf::Uint8* ColorBuffer, *d_ColorBuffer;

	ColorBuffer = new sf::Uint8[WIDTH * HEIGHT * 4];

	for(int i = 0; i < WIDTH * HEIGHT * 4; i++)
	ColorBuffer[i] = 0;

	hipMalloc(&d_ColorBuffer, sizeof(sf::Uint8) * WIDTH * HEIGHT * 4);

	
	Vec3f tri[3] = { Vec3f(0,0,2),Vec3f(0,200,2),Vec3f(200,0,2) };

	while (window.isOpen()) {

		while (window.pollEvent(evnt)) {
			switch (evnt.type) {
			case sf::Event::Closed:
				window.close();
				break;
			case sf::Event::KeyPressed:
				switch (evnt.key.code) {
				case sf::Keyboard::Escape:
					window.close();
					break;
				}
				break;
			case sf::Event::Resized:
				if (window.getSize().x < 600) {
					window.setSize(sf::Vector2u(INITIALWIDTH, HEIGHT));
					break;
				}
				if (window.getSize().y < 600) {
					window.setSize(sf::Vector2u(WIDTH, INITIALHEIGHT));
					break;
				}


				WIDTH = window.getSize().x;
				HEIGHT = window.getSize().y;
				hipFree(ColorBuffer);
				hipMalloc(&ColorBuffer, sizeof(sf::Uint8) * WIDTH * HEIGHT * 4);

					break;
			}
		}

		sf::Clock clock;
		//render

		Render <<<1, 1 >> >(d_ColorBuffer, WIDTH * HEIGHT);
		
		hipMemcpy(ColorBuffer, d_ColorBuffer, sizeof(sf::Uint8) * WIDTH * HEIGHT * 4, hipMemcpyDeviceToHost);
		
		//push render to screen
		screen.update(ColorBuffer);

		window.draw(mSprite);

		window.display();

		std::cout << clock.restart().asMilliseconds() << std::endl;
	}
	hipFree(d_ColorBuffer);
	return 0;
}

